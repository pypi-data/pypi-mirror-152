
#include <hip/hip_runtime.h>
extern "C" __global__
void hitsearch_float64(const int n, const double* spectrum, const double threshold, const double drift_rate,
        double* maxsnr, double* maxdrift, unsigned int* tot_hits, const float median, const float stddev) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    int count = 0;
    for (int i = index; i < n; i += stride) {
        const double bin = (spectrum[i] - median) / stddev;
        if (bin > threshold) {
            count++;
            if (bin > maxsnr[i]) {
                maxsnr[i] = bin;
                maxdrift[i] = drift_rate;
            }
        }
    }
    atomicAdd(&tot_hits[0], count);
}

extern "C" __global__
void hitsearch_float32(const int n, const float* spectrum, const double threshold, const double drift_rate,
        float* maxsnr, float* maxdrift, unsigned int* tot_hits, const float median, const float stddev) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    int count = 0;
    for (int i = index; i < n; i += stride) {
        const double bin = (spectrum[i] - median) / stddev;
        if (bin > threshold) {
            count++;
            if (bin > maxsnr[i]) {
                maxsnr[i] = bin;
                maxdrift[i] = drift_rate;
            }
        }
    }
    atomicAdd(&tot_hits[0], count);
}
